#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2021 CERN
// SPDX-License-Identifier: Apache-2.0

#include "example13.cuh"

#include <AdePT/BVHNavigator.h>

#include <CopCore/PhysicalConstants.h>

#include <G4HepEmGammaManager.hh>
#include <G4HepEmGammaTrack.hh>
#include <G4HepEmTrack.hh>
#include <G4HepEmGammaInteractionCompton.hh>
#include <G4HepEmGammaInteractionConversion.hh>
#include <G4HepEmGammaInteractionPhotoelectric.hh>
// Pull in implementation.
#include <G4HepEmGammaManager.icc>
#include <G4HepEmGammaInteractionCompton.icc>
#include <G4HepEmGammaInteractionConversion.icc>
#include <G4HepEmGammaInteractionPhotoelectric.icc>

__device__
void ComputePhysicsStepLimit(Track &track)
{
  int id = track.navState.Top()->GetLogicalVolume()->id();

  G4HepEmTrack* t = track.gammaTrack.GetTrack();

  t->SetEKin(track.energy);
  t->SetMCIndex(MCIndex[id]);

  for (int ip = 0; ip < 3; ++ip)
    if (t->GetNumIALeft(ip) <= 0)
      t->SetNumIALeft(-std::log(track.Uniform()), ip);

  G4HepEmGammaManager::HowFar(&g4HepEmData, &g4HepEmPars, &track.gammaTrack);
}

__global__ void ComputePhysicsStepLimit(Track *gammas, const adept::MParray *active)
{
  int activeSize = active->size();
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < activeSize; i += blockDim.x * gridDim.x)
    ComputePhysicsStepLimit(gammas[(*active)[i]]);
}

__device__
bool ComputeGeometryStepAndPropagate(Track &track)
{
#ifdef VECGEOM_FLOAT_PRECISION
  const Precision kPush = 10 * vecgeom::kTolerance;
#else
  const Precision kPush = 0.;
#endif
  vecgeom::NavStateIndex nextState;
  G4HepEmTrack* theTrack = track.gammaTrack.GetTrack();

  double StepLength = BVHNavigator::ComputeStepAndNextVolume(track.pos, track.dir,
    theTrack->GetGStepLength(), track.navState, nextState, kPush);

  track.pos += StepLength * track.dir;

  // Propagate information from geometrical step to G4HepEm.
  theTrack->SetGStepLength(StepLength);
  theTrack->SetOnBoundary(nextState.IsOnBoundary());
  G4HepEmGammaManager::UpdateNumIALeft(theTrack);

  // Relocate track if necessary, else set boundary state to propagate
  // information correctly to secondaries and the next step
  track.navState.SetBoundaryState(nextState.IsOnBoundary());

  if (nextState.IsOnBoundary() && nextState.Top()) {
    BVHNavigator::RelocateToNextVolume(track.pos, track.dir, nextState);
    track.navState = nextState;
  }

  return nextState.Top() != nullptr; /* returns if particle is still inside world */
}

__global__ void ComputeGeometryStepAndPropagate(Track *gammas, const adept::MParray *active,
                                                adept::MParray *activeQueue, GlobalScoring *globalScoring)
{
  int activeSize = active->size();
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < activeSize; i += blockDim.x * gridDim.x) {
    int slot      = (*active)[i];
    Track &currentTrack = gammas[slot];

    bool inWorld = ComputeGeometryStepAndPropagate(currentTrack);
    bool onBoundary = currentTrack.navState.IsOnBoundary();

    currentTrack.done = onBoundary && inWorld;

    if (onBoundary) {
      // For now, just count that we hit something.
      atomicAdd(&globalScoring->hits, 1);

      // Kill the particle if it left the world.
      if (inWorld)
        activeQueue->push_back(slot);
    }

    atomicAdd(&globalScoring->neutralSteps, 1);
  }
}

__device__ bool GammaConversion(Track &track, Secondaries &secondaries,
                                GlobalScoring *globalScoring, ScoringPerVolume *)
{
  // Return if energy is below threshold
  if (track.energy < 2 * copcore::units::kElectronMassC2)
    return false;

  double logEnergy = std::log(track.energy);
  double elKinEnergy, posKinEnergy;

  RanluxppDoubleEngine rnge(&track.rngState);
  G4HepEmTrack* theTrack = track.gammaTrack.GetTrack();

  G4HepEmGammaInteractionConversion::SampleKinEnergies(&g4HepEmData, track.energy, logEnergy, theTrack->GetMCIndex(), elKinEnergy,
      posKinEnergy, &rnge);

  double dirPrimary[] = {track.dir.x(), track.dir.y(), track.dir.z()};
  double dirSecondaryEl[3], dirSecondaryPos[3];
  G4HepEmGammaInteractionConversion::SampleDirections(dirPrimary, dirSecondaryEl, dirSecondaryPos, elKinEnergy,
      posKinEnergy, &rnge);

  Track &electron = secondaries.electrons.NextTrack();
  Track &positron = secondaries.positrons.NextTrack();

  electron.InitAsSecondary(/*parent=*/track);
  electron.rngState = track.rngState.Branch();
  electron.energy   = elKinEnergy;
  electron.dir.Set(dirSecondaryEl[0], dirSecondaryEl[1], dirSecondaryEl[2]);

  positron.InitAsSecondary(/*parent=*/track);
  // Reuse the RNG state of the dying track.
  positron.rngState = track.rngState;
  positron.energy   = posKinEnergy;
  positron.dir.Set(dirSecondaryPos[0], dirSecondaryPos[1], dirSecondaryPos[2]);

  atomicAdd(&globalScoring->numElectrons, 1);
  atomicAdd(&globalScoring->numPositrons, 1);

  return true;
}

__device__ bool ComptonScattering(Track &track, Secondaries &secondaries,
                                  GlobalScoring *globalScoring, ScoringPerVolume *scoringPerVolume)
{
  constexpr double LowEnergyThreshold = 100 * copcore::units::eV;

  if (track.energy < LowEnergyThreshold)
    return false;

  RanluxppDoubleEngine rnge(&track.rngState);
  int volumeID = track.navState.Top()->id();

  const double origDirPrimary[] = {track.dir.x(), track.dir.y(), track.dir.z()};
  double dirPrimary[3];
  const double newEnergyGamma =
    G4HepEmGammaInteractionCompton::SamplePhotonEnergyAndDirection(track.energy, dirPrimary, origDirPrimary, &rnge);
  vecgeom::Vector3D<double> newDirGamma(dirPrimary[0], dirPrimary[1], dirPrimary[2]);

  const double energyEl = track.energy - newEnergyGamma;

  if (energyEl > LowEnergyThreshold) {
    // Create a secondary electron and sample/compute directions.
    Track &electron = secondaries.electrons.NextTrack();
    atomicAdd(&globalScoring->numElectrons, 1);

    electron.InitAsSecondary(/*parent=*/track);
    electron.rngState = track.rngState.Branch();
    electron.energy   = energyEl;
    electron.dir      = track.energy * track.dir - newEnergyGamma * newDirGamma;
    electron.dir.Normalize();
  } else {
    atomicAdd(&globalScoring->energyDeposit, energyEl);
    atomicAdd(&scoringPerVolume->energyDeposit[volumeID], energyEl);
  }

  // Check the new gamma energy and deposit if below threshold.
  if (newEnergyGamma > LowEnergyThreshold) {
    track.energy = newEnergyGamma;
    track.dir    = newDirGamma;

    // The current track continues to live.
    return false;
  } else {
    atomicAdd(&globalScoring->energyDeposit, newEnergyGamma);
    atomicAdd(&scoringPerVolume->energyDeposit[volumeID], newEnergyGamma);
    return true;
  }
}

__device__ bool PhotoElectricEffect(Track &track, Secondaries &secondaries,
                                    GlobalScoring *globalScoring, ScoringPerVolume *scoringPerVolume)
{
  // Invoke photoelectric process.
  RanluxppDoubleEngine rnge(&track.rngState);
  const double theLowEnergyThreshold = 1 * copcore::units::eV;

  const double bindingEnergy = G4HepEmGammaInteractionPhotoelectric::SelectElementBindingEnergy(
      &g4HepEmData, track.gammaTrack.GetTrack()->GetMCIndex(), track.gammaTrack.GetPEmxSec(), track.energy, &rnge);

  double edep             = bindingEnergy;
  const double photoElecE = track.energy - edep;
  if (photoElecE > theLowEnergyThreshold) {
    // Create a secondary electron and sample directions.
    Track &electron = secondaries.electrons.NextTrack();
    atomicAdd(&globalScoring->numElectrons, 1);

    double dirGamma[] = {track.dir.x(), track.dir.y(), track.dir.z()};
    double dirPhotoElec[3];
    G4HepEmGammaInteractionPhotoelectric::SamplePhotoElectronDirection(photoElecE, dirGamma, dirPhotoElec, &rnge);

    electron.InitAsSecondary(/*parent=*/track);
    electron.rngState = track.rngState.Branch();
    electron.energy   = photoElecE;
    electron.dir.Set(dirPhotoElec[0], dirPhotoElec[1], dirPhotoElec[2]);
  } else {
    edep = track.energy;
  }
  atomicAdd(&globalScoring->energyDeposit, edep);
  return true;
}

__global__ void ComputeInteraction(Track *gammas, const adept::MParray *active, Secondaries secondaries,
                                adept::MParray *activeQueue, GlobalScoring *globalScoring,
                                ScoringPerVolume *scoringPerVolume)
{
  int activeSize = active->size();
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < activeSize; i += blockDim.x * gridDim.x) {
    const int slot = (*active)[i];
    Track &currentTrack = gammas[slot];

    // Skip tracks that have hit a boundary
    if (currentTrack.done)
      continue;

    G4HepEmTrack* theTrack = currentTrack.gammaTrack.GetTrack();
    int winnerProcessIndex = theTrack->GetWinnerProcessIndex();

    // Skip tracks that have no active discrete process
    if (winnerProcessIndex < 0) {
      activeQueue->push_back(slot);
      continue;
    }

    // Reset number of interaction left. It will be resampled in the next iteration.
    theTrack->SetNumIALeft(-1, theTrack->GetWinnerProcessIndex());

    // Perform the discrete interaction.
    switch (winnerProcessIndex) {
    case 0: {
      if (!GammaConversion(currentTrack, secondaries, globalScoring, scoringPerVolume))
        activeQueue->push_back(slot);
      break;
    }
    case 1: {
      if (!ComptonScattering(currentTrack, secondaries, globalScoring, scoringPerVolume))
        activeQueue->push_back(slot);
      break;
    }
    case 2: {
      PhotoElectricEffect(currentTrack, secondaries, globalScoring, scoringPerVolume);
      break;
    }
    }
  }
}

__global__ void TransportGammas(Track *gammas, const adept::MParray *active, Secondaries secondaries,
                                adept::MParray *activeQueue, GlobalScoring *globalScoring,
                                ScoringPerVolume *scoringPerVolume)
{
  int activeSize = active->size();
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < activeSize; i += blockDim.x * gridDim.x) {
    const int slot      = (*active)[i];
    Track &currentTrack = gammas[slot];

    ComputePhysicsStepLimit(currentTrack);

    G4HepEmTrack* theTrack = currentTrack.gammaTrack.GetTrack();
    int winnerProcessIndex = theTrack->GetWinnerProcessIndex();

    bool inWorld = ComputeGeometryStepAndPropagate(currentTrack);
    atomicAdd(&globalScoring->neutralSteps, 1);

    if (theTrack->GetOnBoundary()) {
      // For now, just count that we hit something.
      atomicAdd(&globalScoring->hits, 1);

      // Kill the particle if it left the world.
      if (inWorld)
        activeQueue->push_back(slot);

      continue;
    }

    // No discrete process, move on.
    if (winnerProcessIndex < 0) {
      activeQueue->push_back(slot);
      continue;
    }

    // Reset number of interaction left for the winner discrete process.
    // (Will be resampled in the next iteration.)
    theTrack->SetNumIALeft(-1, theTrack->GetWinnerProcessIndex());

    // Perform the discrete interaction.
    switch (winnerProcessIndex) {
    case 0: {
      if (!GammaConversion(currentTrack, secondaries, globalScoring, scoringPerVolume))
        activeQueue->push_back(slot);
      break;
    }
    case 1: {
      if (!ComptonScattering(currentTrack, secondaries, globalScoring, scoringPerVolume))
        activeQueue->push_back(slot);
      break;
    }
    case 2: {
      PhotoElectricEffect(currentTrack, secondaries, globalScoring, scoringPerVolume);
      break;
    }
    }
  }
}
